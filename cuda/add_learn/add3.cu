
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

const double EPLISION = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

__global__ void add(const double *x, const double *y, double *z, const int N);

__device__ double add1_device(const double x, const double y);
__global__ void add1(const double *x, const double *y, double *z, const int N);

__device__ void add2_device(const double x, const double y, double *z);
__global__ void add2(const double *x, const double *y, double *z, const int N);

__device__ void add3_device(const double x, const double y, double &z);
__global__ void add3(const double *x, const double *y, double *z, const int N);

void check(const double *z, const int N);

int main()
{
    // 这里的N超过1e7就会报错，我认为这应该与gpu的计算能力相关
    //
    const int N = 1e7;
    const int M = sizeof(double) * N;
    double *h_x = new double[N];
    double *h_y = new double[N];
    double *h_z = new double[N];

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    double *d_x, *d_y, *d_z;

    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int gride_size = (N % block_size == 0 ? N / block_size
                                                : N / block_size + 1);
    // add<<<gride_size, block_size>>>(d_x, d_y, d_z, N);

    // add1<<<gride_size, block_size>>>(d_x, d_y, d_z, N);
    // add2<<<gride_size, block_size>>>(d_x, d_y, d_z, N);
    add3<<<gride_size, block_size>>>(d_x, d_y, d_z, N);

    hipDeviceSynchronize();

    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);
    delete[] h_x;
    delete[] h_y;
    delete[] h_z;
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}

__global__ void add(const double *x, const double *y, double *z, const int N)
{
    // get index
    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    // base index to compute
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
    //printf("z[%d]=%f\n", n,z[n]);
}

// 有返回值的设备函数
__device__ double add1_device(const double x, const double y)
{
    return x + y;
}

__global__ void add1(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = add1_device(x[n], y[n]);
    }
}

// 用指针的设备函数
__device__ void add2_device(const double x, const double y, double *z)
{
    *z = x + y;
}

__global__ void add2(const double *x, const double *y, double *z, const int N)
{

    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        add2_device(x[n], y[n], &z[n]);
    }
}

// 用引用的设备函数
__device__ void add3_device(const double x, const double y, double &z)
{
    z = x + y;
}

__global__ void add3(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        add3_device(x[n], y[n], z[n]);
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        // printf("%f\n",z[n]);
        if (fabs(z[n] - c) > EPLISION)
        {
            printf("z[%d]=%f,c=%f\n", n, z[n], c);
            has_error = true;
            break;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

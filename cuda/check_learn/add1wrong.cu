#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "error.cuh"

const double EPLISION = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

__global__ void add(const double *x, const double *y, double *z, const int N);
void check(const double *z, const int N);

int main()
{
    // 这里的N超过1e7就会报错，我认为这应该与gpu的计算能力相关
    //
    const int N = 1e8;
    const int M = sizeof(double) * N;
    double *h_x = new double[N];
    double *h_y = new double[N];
    double *h_z = new double[N];

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    double *d_x, *d_y, *d_z;

    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int gride_size = (N % block_size == 0 ? N / block_size
                                                : N / block_size + 1);
    add<<<gride_size, block_size>>>(d_x, d_y, d_z, N);
    hipDeviceSynchronize();

    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);
    delete[] h_x;
    delete[] h_y;
    delete[] h_z;
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    return 0;
}

__global__ void add(const double *x, const double *y, double *z, const int N)
{
    // get index
    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    // base index to compute
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
    //printf("z[%d]=%f\n", n,z[n]);
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        // printf("%f\n",z[n]);
        if (fabs(z[n] - c) > EPLISION)
        {
            printf("z[%d]=%f,c=%f\n", n, z[n], c);
            has_error = true;
            break;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}
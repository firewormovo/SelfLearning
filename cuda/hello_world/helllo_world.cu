
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu(){
    printf("Hello world from gpu!\n");
}

int main(){
    hello_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu() { printf("hello world from gpu\n"); }

int main() {
  hello_from_gpu<<<2, 3>>>();
  hipDeviceSynchronize();
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu(){
    printf("hello world from gpu\n");
}

void __global__ hello_from_other_gpu(){
    printf("hello from other gpu\n");
}

int main(void){
    hello_from_gpu<<<1,1>>>();
    hello_from_other_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
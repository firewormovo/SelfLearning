
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("girdm.x: %d ,blockdim.x: %d \n",gridDim.x - 1,blockDim.x - 1);
    printf("Hello from block %d and thread %d\n", bid, tid);
}

int main(){
    hello_from_gpu<<<5,6>>>();
    hipDeviceSynchronize();
    return 0;
}
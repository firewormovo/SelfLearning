
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu() {
  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;
  printf("hello world from gpu , block id is %d ,thread id is %d \n", block_id,
         thread_id);
}

int main() {
  hello_from_gpu<<<3, 12>>>();
  hipDeviceSynchronize();
  return 0;
}